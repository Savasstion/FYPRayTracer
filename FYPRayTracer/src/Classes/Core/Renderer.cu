#include "hip/hip_runtime.h"
#include "Renderer.h"
#include "RendererCUDA.cuh"
#include "../../Utility/ColorUtils.cuh"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <>
#include <iostream>
#include <glm/gtx/compatibility.hpp>

Scene_GPU* RendererGPU::d_currentScene = nullptr;

void Renderer::Render(Scene& scene, Camera& camera)
{
    m_ActiveScene = &scene;
    m_ActiveCamera = &camera;
    
    uint32_t width  = m_FinalRenderImage->GetWidth();
    uint32_t height = m_FinalRenderImage->GetHeight();
    size_t pixelCount = width * height;
    
    constexpr size_t vec4Size = sizeof(glm::vec4);
    constexpr size_t uint32Size = sizeof(uint32_t);

    // Allocate device buffers for accumulation and output image
    glm::vec4* d_accumulationData = nullptr;
    uint32_t* d_renderImageData = nullptr;

    hipError_t err;
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "cuda shit error: " << hipGetErrorString(err) << std::endl;
    }
    
    err = hipMalloc((void**)&d_accumulationData, pixelCount * vec4Size);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc error: " << hipGetErrorString(err) << std::endl;
    }
    
    err = hipMalloc((void**)&d_renderImageData, pixelCount * uint32Size);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc error: " << hipGetErrorString(err) << std::endl;
    }

    // Initialize or copy accumulation buffer
    if (m_FrameIndex == 1)
        err = hipMemset(d_accumulationData, 0, pixelCount * vec4Size);
    else
        err = hipMemcpy(d_accumulationData, m_AccumulationData, pixelCount * vec4Size, hipMemcpyHostToDevice);
    
    if (err != hipSuccess) {
        std::cerr << "cuda copy error: " << hipGetErrorString(err) << std::endl;
    }
    
    // Allocate device versions
    Scene_GPU* d_sceneGPU = RendererGPU::d_currentScene;
    if(isSceneUpdated)
    {
        if(d_sceneGPU != nullptr)
            FreeSceneGPU(d_sceneGPU);
        d_sceneGPU = SceneToGPU(scene);
        RendererGPU::d_currentScene = d_sceneGPU;
        isSceneUpdated = false;
    }
    Camera_GPU* d_cameraGPU = CameraToGPU(camera);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "cuda shit error: " << hipGetErrorString(err) << std::endl;
    }
    
    
    // Configure kernel launch
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(
        (width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    //  Do per pixel rendering task in parallel for all pixels
    RenderKernel<<<numBlocks, threadsPerBlock>>>(
        d_accumulationData,
        d_renderImageData,
        width,
        height,
        m_FrameIndex,
        m_Settings,
        d_sceneGPU,
        d_cameraGPU
    );
    
     err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "hipDeviceSynchronize error: " << hipGetErrorString(err) << std::endl;
    }
    
    // Copy results back
    err = hipMemcpy(m_AccumulationData, d_accumulationData, pixelCount * vec4Size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy error: " << hipGetErrorString(err) << std::endl;
    }
    
    err = hipMemcpy(m_RenderImageData, d_renderImageData, pixelCount * uint32Size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy error: " << hipGetErrorString(err) << std::endl;
    }
    
    m_FinalRenderImage->SetData(m_RenderImageData);

    if (m_Settings.toAccumulate)
        m_FrameIndex++;
    else
        m_FrameIndex = 1;


    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "cuda shit error: " << hipGetErrorString(err) << std::endl;
    }
    
    // Free device memory
    err = hipFree(d_accumulationData);
    if (err != hipSuccess) {
        std::cerr << "hipFree error: " << hipGetErrorString(err) << std::endl;
    }
    
    err = hipFree(d_renderImageData);
    if (err != hipSuccess) {
        std::cerr << "hipFree error: " << hipGetErrorString(err) << std::endl;
    }
    
    FreeCameraGPU(d_cameraGPU);
    
}


__host__ __device__ RayHitPayload RendererGPU::TraceRay(const Ray& ray, const Scene_GPU* activeScene)
{
    if (activeScene->triangleCount == 0) return Miss(ray);

    float closestHitDistance = FLT_MAX;
    int closestTriangle = -1;
    float closestU = 0.0f;
    float closestV = 0.0f;

    //TLAS - BLAS Traversal
     BVH* tlas = activeScene->tlas;
    
     const int TLAS_STACK_SIZE = 256;
     int tlasStack[TLAS_STACK_SIZE];
     int tlasStackTop = 0;
    
     const int BLAS_STACK_SIZE = 1024;
     int blasStack[BLAS_STACK_SIZE];
    
     tlasStack[tlasStackTop++] = static_cast<int>(tlas->rootIndex);
    
     while (tlasStackTop > 0)
     {
         int nodeIndex = tlasStack[--tlasStackTop];
    
         const BVH::Node& node = tlas->nodes[nodeIndex];
         if (!IntersectRayAABB(ray, node.box)) continue;
    
         if (node.isLeaf)
         {
             size_t meshIndex = node.objectIndex;
             
             BVH* blas = activeScene->meshes[meshIndex].blas;
             
             int blasStackTop = 0;
             blasStack[blasStackTop++] = static_cast<int>(blas->rootIndex);
             
             while (blasStackTop > 0)
             {
                 int bnodeIndex = blasStack[--blasStackTop];
    
                 const BVH::Node& bnode = blas->nodes[bnodeIndex];
                 if (!IntersectRayAABB(ray, bnode.box)) continue;
    
                 if (bnode.isLeaf)
                 {
                     size_t triangleIndex = bnode.objectIndex;

                     const Triangle& tri = activeScene->triangles[triangleIndex];
                     const glm::vec3& v0 = activeScene->worldVertices[tri.v0].position;
                     const glm::vec3& v1 = activeScene->worldVertices[tri.v1].position;
                     const glm::vec3& v2 = activeScene->worldVertices[tri.v2].position;

                     // Möller–Trumbore intersection algorithm
                     glm::vec3 edge1 = v1 - v0;
                     glm::vec3 edge2 = v2 - v0;
                     glm::vec3 h = glm::cross(ray.direction, edge2);
                     float a = glm::dot(edge1, h);
                     // if (fabsf(a) < 1e-8f) continue;
    
                     float f = 1.0f / a;
                     glm::vec3 s = ray.origin - v0;
                     float u = f * glm::dot(s, h);
                     if (u < 0.0f || u > 1.0f) continue;
    
                     glm::vec3 q = glm::cross(s, edge1);
                     float v = f * glm::dot(ray.direction, q);
                     if (v < 0.0f || (u + v) > 1.0f) continue;
    
                     float t = f * glm::dot(edge2, q);
    
                     if (t > 0.0001f && t < closestHitDistance)
                     {
                         closestHitDistance = t;
                         closestTriangle = static_cast<int>(triangleIndex);
                         closestU = u;
                         closestV = v;
                     }
                 }
                 else
                 {
                     if (bnode.child1 != static_cast<size_t>(-1) && blasStackTop < BLAS_STACK_SIZE)
                         blasStack[blasStackTop++] = static_cast<int>(bnode.child1);
                     if (bnode.child2 != static_cast<size_t>(-1) && blasStackTop < BLAS_STACK_SIZE)
                         blasStack[blasStackTop++] = static_cast<int>(bnode.child2);
                 }
             }
         }
         else
         {
             if (node.child1 != static_cast<size_t>(-1) && tlasStackTop < TLAS_STACK_SIZE)
                 tlasStack[tlasStackTop++] = static_cast<int>(node.child1);
             if (node.child2 != static_cast<size_t>(-1) && tlasStackTop < TLAS_STACK_SIZE)
                 tlasStack[tlasStackTop++] = static_cast<int>(node.child2);
         }
     }

    if (closestTriangle < 0)
        return Miss(ray);

    return ClosestHit(ray, closestHitDistance, closestTriangle, closestU, closestV, activeScene);
}


 //  PURE BRUTE-FORCE
__host__ __device__ glm::vec4 RendererGPU::PerPixel_BruteForce(
    uint32_t x, uint32_t y,
    uint8_t maxBounces, uint8_t sampleCount,
    uint32_t frameIndex, const RenderingSettings& settings,
    const Scene_GPU* activeScene, const Camera_GPU* activeCamera,
    uint32_t imageWidth)
{
    //  PURE BRUTE-FORCE, EVERY FRAME ONLY CHOOSE ONE RANDOM PATH TO FOLLOW
    //  LIKE A OFFLINE-RENDERER, KEEPS TRACK OF SUM AND THEN AVERAGE THE PIXEL'S COLOR OVER TIME TO EVENTUALLY FORM A PHYSICALLY-ACCURATE IMAGE (keyword : "eventually")
    
    //  all directions within a hemisphere are all equally likely to be sampled
    
    uint32_t seed = x + y * imageWidth;
    seed *= frameIndex;

    glm::vec3 radiance{0.0f};   // Final color accumulated from all samples

    // == PRIMARY RAY ==
    Ray primaryRay;
    primaryRay.origin = activeCamera->position;
    primaryRay.direction = activeCamera->rayDirections[x + y * imageWidth];

    RayHitPayload primaryPayload = TraceRay(primaryRay, activeScene);

    // Hit sky immediately
    if (primaryPayload.hitDistance < 0.0f)
        return glm::vec4(settings.skyColor, 1.0f);

    const Triangle& hitTri = activeScene->triangles[primaryPayload.objectIndex];
    const Material& hitMaterial = activeScene->materials[hitTri.materialIndex];

    // Hit emissive object immediately
    if (glm::length(hitMaterial.GetEmission()) > 0.0f)
        return glm::vec4(hitMaterial.GetEmission(), 1.0f);
    
    glm::vec3 sampleThroughput{1.0f};
    Ray sampleRay;
    RayHitPayload samplePayload = primaryPayload;

    // Sample initial direction from first hit
    glm::vec3 newDir = MathUtils::UniformSampleHemisphere(primaryPayload.worldNormal, seed);

    glm::vec3 brdf = MathUtils::CalculateBRDF(
        primaryPayload.worldNormal,
        -primaryRay.direction,
        newDir,
        hitMaterial.albedo,
        hitMaterial.metallic,
        hitMaterial.roughness
    );

    float cosTheta = glm::max(glm::dot(newDir, primaryPayload.worldNormal), 0.0f);  //  Geometry Term
    float pdf = MathUtils::UniformHemispherePDF();
    sampleThroughput *= brdf * cosTheta / pdf; // Rendering equation core

    sampleRay.origin = primaryPayload.worldPosition + primaryPayload.worldNormal * 1e-12f;
    sampleRay.direction = newDir;

    // Trace the path for maxBounces
    for (int bounce = 0; bounce < maxBounces; bounce++)
    {
        seed += 31 * bounce;

        samplePayload = TraceRay(sampleRay, activeScene);

        // Hit sky
        if (samplePayload.hitDistance < 0.0f)
        {
            radiance += sampleThroughput * settings.skyColor;
            break;
        }

        const Triangle& tri = activeScene->triangles[samplePayload.objectIndex];
        const Material& material = activeScene->materials[tri.materialIndex];

        // Hit emissive light
        glm::vec3 emission = material.GetEmission();
        if (glm::length(emission) > 0.0f)
        {
            radiance += sampleThroughput * emission;
            break;
        }

        // Sample next direction
        glm::vec3 bounceDir = MathUtils::UniformSampleHemisphere(samplePayload.worldNormal, seed);
            
        glm::vec3 bounceBrdf = MathUtils::CalculateBRDF(
        samplePayload.worldNormal,
        -sampleRay.direction,
        bounceDir,
        material.albedo,
        material.metallic,
        material.roughness
        );

        float bounceCosTheta = glm::max(glm::dot(bounceDir, samplePayload.worldNormal), 0.0f);
        float bouncePdf = MathUtils::UniformHemispherePDF();
        sampleThroughput *= bounceBrdf * bounceCosTheta / bouncePdf;

        sampleRay.origin = samplePayload.worldPosition + samplePayload.worldNormal * 1e-12f;
        sampleRay.direction = bounceDir;
    }
    
    return glm::vec4(radiance, 1.0f);
}

 //  UNIFORM SAMPLING
__host__ __device__ glm::vec4 RendererGPU::PerPixel_UniformSampling(
    uint32_t x, uint32_t y,
    uint8_t maxBounces, uint8_t sampleCount,
    uint32_t frameIndex, const RenderingSettings& settings,
    const Scene_GPU* activeScene, const Camera_GPU* activeCamera,
    uint32_t imageWidth)
{
    //  all directions within a hemisphere are all equally likely to be sampled
    
    uint32_t seed = x + y * imageWidth;
    seed *= frameIndex;

    glm::vec3 radiance{0.0f};   // Final color accumulated from all samples

    // == PRIMARY RAY ==
    Ray primaryRay;
    primaryRay.origin = activeCamera->position;
    primaryRay.direction = activeCamera->rayDirections[x + y * imageWidth];

    RayHitPayload primaryPayload = TraceRay(primaryRay, activeScene);

    // Hit sky immediately
    if (primaryPayload.hitDistance < 0.0f)
        return glm::vec4(settings.skyColor, 1.0f);

    const Triangle& hitTri = activeScene->triangles[primaryPayload.objectIndex];
    const Material& hitMaterial = activeScene->materials[hitTri.materialIndex];

    // Hit emissive object immediately
    if (glm::length(hitMaterial.GetEmission()) > 0.0f)
        return glm::vec4(hitMaterial.GetEmission(), 1.0f);

    // == SAMPLE MULTIPLE LIGHT PATHS FROM FIRST HIT ==
    for (int sampleIndex = 0; sampleIndex < sampleCount; sampleIndex++)
    {
        seed += (sampleIndex + 1) * 27;
        glm::vec3 sampleThroughput{1.0f};
        Ray sampleRay;
        RayHitPayload samplePayload = primaryPayload;

        // Sample initial direction from first hit
        glm::vec3 newDir = MathUtils::UniformSampleHemisphere(primaryPayload.worldNormal, seed);

        glm::vec3 brdf = MathUtils::CalculateBRDF(
            primaryPayload.worldNormal,
            -primaryRay.direction,
            newDir,
            hitMaterial.albedo,
            hitMaterial.metallic,
            hitMaterial.roughness
        );

        float cosTheta = glm::max(glm::dot(newDir, primaryPayload.worldNormal), 0.0f);  //  Geometry Term
        float pdf = MathUtils::UniformHemispherePDF();
        sampleThroughput *= brdf * cosTheta / pdf; // Rendering equation core

        sampleRay.origin = primaryPayload.worldPosition + primaryPayload.worldNormal * 1e-12f;
        sampleRay.direction = newDir;

        // Trace the path for maxBounces
        for (int bounce = 0; bounce < maxBounces; bounce++)
        {
            seed += sampleIndex + 31 * bounce;

            samplePayload = TraceRay(sampleRay, activeScene);

            // Hit sky
            if (samplePayload.hitDistance < 0.0f)
            {
                radiance += sampleThroughput * settings.skyColor;
                break;
            }

            const Triangle& tri = activeScene->triangles[samplePayload.objectIndex];
            const Material& material = activeScene->materials[tri.materialIndex];

            // Hit emissive light
            glm::vec3 emission = material.GetEmission();
            if (glm::length(emission) > 0.0f)
            {
                radiance += sampleThroughput * emission;
                break;
            }

            // Sample next direction
            glm::vec3 bounceDir = MathUtils::UniformSampleHemisphere(samplePayload.worldNormal, seed);
            
            glm::vec3 bounceBrdf = MathUtils::CalculateBRDF(
                samplePayload.worldNormal,
                -sampleRay.direction,
                bounceDir,
                material.albedo,
                material.metallic,
                material.roughness
            );

            float bounceCosTheta = glm::max(glm::dot(bounceDir, samplePayload.worldNormal), 0.0f);
            float bouncePdf = MathUtils::UniformHemispherePDF();
            sampleThroughput *= bounceBrdf * bounceCosTheta / bouncePdf;

            sampleRay.origin = samplePayload.worldPosition + samplePayload.worldNormal * 1e-12f;
            sampleRay.direction = bounceDir;
        }
    }

    radiance /= float(sampleCount); // Average across all sampled paths
    return glm::vec4(radiance, 1.0f);
}

 //  COSINE-WEIGHTED SAMPLING
__host__ __device__ glm::vec4 RendererGPU::PerPixel_CosineWeightedSampling(
    uint32_t x, uint32_t y,
    uint8_t maxBounces, uint8_t sampleCount,
    uint32_t frameIndex, const RenderingSettings& settings,
    const Scene_GPU* activeScene, const Camera_GPU* activeCamera,
    uint32_t imageWidth)
{
    //  a kind of BRDF sampling method that makes it so that shallower angles are less likely to be sampled as they likely contribute less light
    
    uint32_t seed = x + y * imageWidth;
    seed *= frameIndex;

    glm::vec3 radiance{0.0f};   // Final color accumulated from all samples

    // == PRIMARY RAY ==
    Ray primaryRay;
    primaryRay.origin = activeCamera->position;
    primaryRay.direction = activeCamera->rayDirections[x + y * imageWidth];

    RayHitPayload primaryPayload = TraceRay(primaryRay, activeScene);

    // Hit sky immediately
    if (primaryPayload.hitDistance < 0.0f)
        return glm::vec4(settings.skyColor, 1.0f);

    const Triangle& hitTri = activeScene->triangles[primaryPayload.objectIndex];
    const Material& hitMaterial = activeScene->materials[hitTri.materialIndex];

    // Hit emissive object immediately
    if (glm::length(hitMaterial.GetEmission()) > 0.0f)
        return glm::vec4(hitMaterial.GetEmission(), 1.0f);

    // == SAMPLE MULTIPLE LIGHT PATHS FROM FIRST HIT ==
    for (int sampleIndex = 0; sampleIndex < sampleCount; sampleIndex++)
    {
        seed += (sampleIndex + 1) * 27;
        glm::vec3 sampleThroughput{1.0f};
        Ray sampleRay;
        RayHitPayload samplePayload = primaryPayload;

        // Sample initial direction from first hit
        glm::vec3 newDir = MathUtils::CosineSampleHemisphere(primaryPayload.worldNormal, seed);

        glm::vec3 brdf = MathUtils::CalculateBRDF(
            primaryPayload.worldNormal,
            -primaryRay.direction,
            newDir,
            hitMaterial.albedo,
            hitMaterial.metallic,
            hitMaterial.roughness
        );

        float cosTheta = glm::max(glm::dot(newDir, primaryPayload.worldNormal), 0.0f);  //  Geometry Term
        float pdf = MathUtils::CosineHemispherePDF(cosTheta);
        sampleThroughput *= brdf * cosTheta / pdf; // Rendering equation core

        sampleRay.origin = primaryPayload.worldPosition + primaryPayload.worldNormal * 1e-12f;
        sampleRay.direction = newDir;

        // Trace the path for maxBounces
        for (int bounce = 0; bounce < maxBounces; bounce++)
        {
            seed += sampleIndex + 31 * bounce;

            samplePayload = TraceRay(sampleRay, activeScene);

            // Hit sky
            if (samplePayload.hitDistance < 0.0f)
            {
                radiance += sampleThroughput * settings.skyColor;
                break;
            }

            const Triangle& tri = activeScene->triangles[samplePayload.objectIndex];
            const Material& material = activeScene->materials[tri.materialIndex];

            // Hit emissive light
            glm::vec3 emission = material.GetEmission();
            if (glm::length(emission) > 0.0f)
            {
                radiance += sampleThroughput * emission;
                break;
            }

            // Sample next direction
            glm::vec3 bounceDir = MathUtils::CosineSampleHemisphere(samplePayload.worldNormal, seed);
            
            glm::vec3 bounceBrdf = MathUtils::CalculateBRDF(
                samplePayload.worldNormal,
                -sampleRay.direction,
                bounceDir,
                material.albedo,
                material.metallic,
                material.roughness
            );

            float bounceCosTheta = glm::max(glm::dot(bounceDir, samplePayload.worldNormal), 0.0f);
            float bouncePdf = MathUtils::CosineHemispherePDF(bounceCosTheta);
            sampleThroughput *= bounceBrdf * bounceCosTheta / bouncePdf;

            sampleRay.origin = samplePayload.worldPosition + samplePayload.worldNormal * 1e-12f;
            sampleRay.direction = bounceDir;
        }
    }

    radiance /= float(sampleCount); // Average across all sampled paths
    return glm::vec4(radiance, 1.0f);
}

 //  GGX SAMPLING
__host__ __device__ glm::vec4 RendererGPU::PerPixel_GGXSampling(
    uint32_t x, uint32_t y,
    uint8_t maxBounces, uint8_t sampleCount,
    uint32_t frameIndex, const RenderingSettings& settings,
    const Scene_GPU* activeScene, const Camera_GPU* activeCamera,
    uint32_t imageWidth)
{
    //  sample more often towards directions that contribute more to specular lighting
    
    uint32_t seed = x + y * imageWidth;
    seed *= frameIndex;

    glm::vec3 radiance{0.0f};   // Final color accumulated from all samples

    // == PRIMARY RAY ==
    Ray primaryRay;
    primaryRay.origin = activeCamera->position;
    primaryRay.direction = activeCamera->rayDirections[x + y * imageWidth];

    RayHitPayload primaryPayload = TraceRay(primaryRay, activeScene);

    // Hit sky immediately
    if (primaryPayload.hitDistance < 0.0f)
        return glm::vec4(settings.skyColor, 1.0f);

    const Triangle& hitTri = activeScene->triangles[primaryPayload.objectIndex];
    const Material& hitMaterial = activeScene->materials[hitTri.materialIndex];

    // Hit emissive object immediately
    if (glm::length(hitMaterial.GetEmission()) > 0.0f)
        return glm::vec4(hitMaterial.GetEmission(), 1.0f);

    // == SAMPLE MULTIPLE LIGHT PATHS FROM FIRST HIT ==
    for (int sampleIndex = 0; sampleIndex < sampleCount; sampleIndex++)
    {
        seed += (sampleIndex + 1) * 27;
        glm::vec3 sampleThroughput{1.0f};
        Ray sampleRay;
        RayHitPayload samplePayload = primaryPayload;

        // Sample initial direction from first hit
        float pdf;
        glm::vec3 newDir = MathUtils::GGXSampleHemisphere(primaryPayload.worldNormal, -primaryRay.direction, hitMaterial.roughness,seed, pdf);

        glm::vec3 brdf = MathUtils::CalculateBRDF(
            primaryPayload.worldNormal,
            -primaryRay.direction,
            newDir,
            hitMaterial.albedo,
            hitMaterial.metallic,
            hitMaterial.roughness
        );

        float cosTheta = glm::max(glm::dot(newDir, primaryPayload.worldNormal), 0.0f);  //  Geometry Term
        sampleThroughput *= brdf * cosTheta / pdf; // Rendering equation core

        sampleRay.origin = primaryPayload.worldPosition + primaryPayload.worldNormal * 1e-12f;
        sampleRay.direction = newDir;

        // Trace the path for maxBounces
        for (int bounce = 0; bounce < maxBounces; bounce++)
        {
            seed += sampleIndex + 31 * bounce;

            samplePayload = TraceRay(sampleRay, activeScene);

            // Hit sky
            if (samplePayload.hitDistance < 0.0f)
            {
                radiance += sampleThroughput * settings.skyColor;
                break;
            }

            const Triangle& tri = activeScene->triangles[samplePayload.objectIndex];
            const Material& material = activeScene->materials[tri.materialIndex];

            // Hit emissive light
            glm::vec3 emission = material.GetEmission();
            if (glm::length(emission) > 0.0f)
            {
                radiance += sampleThroughput * emission;
                break;
            }

            // Sample next direction
            float bouncePdf;
            glm::vec3 bounceDir = MathUtils::GGXSampleHemisphere(samplePayload.worldNormal, -sampleRay.direction, hitMaterial.roughness,seed, bouncePdf);
            
            glm::vec3 bounceBrdf = MathUtils::CalculateBRDF(
                samplePayload.worldNormal,
                -sampleRay.direction,
                bounceDir,
                material.albedo,
                material.metallic,
                material.roughness
            );

            float bounceCosTheta = glm::max(glm::dot(bounceDir, samplePayload.worldNormal), 0.0f);
            sampleThroughput *= bounceBrdf * bounceCosTheta / bouncePdf;

            sampleRay.origin = samplePayload.worldPosition + samplePayload.worldNormal * 1e-12f;
            sampleRay.direction = bounceDir;
        }
    }

    radiance /= float(sampleCount); // Average across all sampled paths
    return glm::vec4(radiance, 1.0f);
}

//  BRDF SAMPLING
__host__ __device__ glm::vec4 RendererGPU::PerPixel_BRDFSampling(
    uint32_t x, uint32_t y,
    uint8_t maxBounces, uint8_t sampleCount,
    uint32_t frameIndex, const RenderingSettings& settings,
    const Scene_GPU* activeScene, const Camera_GPU* activeCamera,
    uint32_t imageWidth)
{
    uint32_t seed = x + y * imageWidth;
    seed *= frameIndex;

    glm::vec3 radiance{0.0f};

    // PRIMARY RAY
    Ray primaryRay;
    primaryRay.origin = activeCamera->position;
    primaryRay.direction = activeCamera->rayDirections[x + y * imageWidth];

    RayHitPayload primaryPayload = TraceRay(primaryRay, activeScene);

    // Miss: hit sky
    if (primaryPayload.hitDistance < 0.0f)
        return glm::vec4(settings.skyColor, 1.0f);

    const Triangle& hitTri = activeScene->triangles[primaryPayload.objectIndex];
    const Material& hitMaterial = activeScene->materials[hitTri.materialIndex];

    // Hit emissive surface
    if (glm::length(hitMaterial.GetEmission()) > 0.0f)
        return glm::vec4(hitMaterial.GetEmission(), 1.0f);

    // MULTI-SAMPLE LOOP
    for (int sampleIndex = 0; sampleIndex < sampleCount; sampleIndex++)
    {
        seed += (sampleIndex + 1) * 27;
        glm::vec3 sampleThroughput{1.0f};
        Ray sampleRay;
        RayHitPayload samplePayload = primaryPayload;

        // Sample initial bounce
        float pdf;
        glm::vec3 newDir = MathUtils::BRDFSampleHemisphere(
            primaryPayload.worldNormal,
            -primaryRay.direction,
            hitMaterial.albedo,
            hitMaterial.metallic,
            hitMaterial.roughness,
            seed,
            pdf
        );

        glm::vec3 brdf = MathUtils::CalculateBRDF(
            primaryPayload.worldNormal,
            -primaryRay.direction,
            newDir,
            hitMaterial.albedo,
            hitMaterial.metallic,
            hitMaterial.roughness
        );

        float cosTheta = glm::max(glm::dot(newDir, primaryPayload.worldNormal), 0.0f);
        sampleThroughput *= brdf * cosTheta / pdf;

        sampleRay.origin = primaryPayload.worldPosition + primaryPayload.worldNormal * 1e-12f;
        sampleRay.direction = newDir;

        // BOUNCE LOOP
        for (int bounce = 0; bounce < maxBounces; bounce++)
        {
            seed += sampleIndex + 31 * bounce;
            samplePayload = TraceRay(sampleRay, activeScene);

            // Miss: hit sky
            if (samplePayload.hitDistance < 0.0f)
            {
                radiance += sampleThroughput * settings.skyColor;
                break;
            }

            const Triangle& tri = activeScene->triangles[samplePayload.objectIndex];
            const Material& material = activeScene->materials[tri.materialIndex];

            // Hit emissive
            glm::vec3 emission = material.GetEmission();
            if (glm::length(emission) > 0.0f)
            {
                radiance += sampleThroughput * emission;
                break;
            }

            // Next bounce
            float bouncePdf;
            glm::vec3 bounceDir = MathUtils::BRDFSampleHemisphere(
                samplePayload.worldNormal,
                -sampleRay.direction,
                material.albedo,
                material.metallic,
                material.roughness,
                seed,
                bouncePdf
            );

            glm::vec3 bounceBrdf = MathUtils::CalculateBRDF(
                samplePayload.worldNormal,
                -sampleRay.direction,
                bounceDir,
                material.albedo,
                material.metallic,
                material.roughness
            );

            float bounceCosTheta = glm::max(glm::dot(bounceDir, samplePayload.worldNormal), 0.0f);
            sampleThroughput *= bounceBrdf * bounceCosTheta / bouncePdf;

            sampleRay.origin = samplePayload.worldPosition + samplePayload.worldNormal * 1e-12f;
            sampleRay.direction = bounceDir;
        }
    }

    radiance /= float(sampleCount);
    return glm::vec4(radiance, 1.0f);
}

//  LIGHT SOURCE SAMPLING
__host__ __device__ glm::vec4 RendererGPU::PerPixel_LightSourceSampling(
    uint32_t x, uint32_t y,
    uint8_t maxBounces, uint8_t sampleCount,
    uint32_t frameIndex, const RenderingSettings& settings,
    const Scene_GPU* activeScene, const Camera_GPU* activeCamera,
    uint32_t imageWidth){
    uint32_t seed = x + y * imageWidth;
    seed *= frameIndex;

    glm::vec3 radiance{0.0f};

    // PRIMARY RAY
    Ray primaryRay;
    primaryRay.origin = activeCamera->position;
    primaryRay.direction = activeCamera->rayDirections[x + y * imageWidth];

    RayHitPayload primaryPayload = TraceRay(primaryRay, activeScene);

    // Miss: hit sky
    if (primaryPayload.hitDistance < 0.0f)
        return glm::vec4(settings.skyColor, 1.0f);

    const Triangle& hitTri = activeScene->triangles[primaryPayload.objectIndex];
    const Material& hitMaterial = activeScene->materials[hitTri.materialIndex];

    // Hit emissive surface
    if (glm::length(hitMaterial.GetEmission()) > 0.0f)
        return glm::vec4(hitMaterial.GetEmission(), 1.0f);

    // MULTI-SAMPLE LOOP
    for (int sampleIndex = 0; sampleIndex < sampleCount; sampleIndex++)
    {
        seed += (sampleIndex + 1) * 27;
        glm::vec3 sampleThroughput{1.0f};
        Ray sampleRay;
        RayHitPayload samplePayload = primaryPayload;

        // sample direction to light source
        LightTree::ShadingPointQuery sp;
        sp.normal = primaryPayload.worldNormal;
        sp.position = primaryPayload.worldPosition;
        LightTree::SampledLight sampledLight = PickLight_TLAS(activeScene->meshes, activeScene->lightTree_tlas, sp, seed);

        //  get emmisive triangle data
        glm::vec3 p0 = activeScene->worldVertices[activeScene->triangles[sampledLight.emitterIndex].v0].position;
        glm::vec3 p1 = activeScene->worldVertices[activeScene->triangles[sampledLight.emitterIndex].v1].position;
        glm::vec3 p2 = activeScene->worldVertices[activeScene->triangles[sampledLight.emitterIndex].v2].position;
        glm::vec3 n0 = activeScene->worldVertices[activeScene->triangles[sampledLight.emitterIndex].v0].normal;
        glm::vec3 n1 = activeScene->worldVertices[activeScene->triangles[sampledLight.emitterIndex].v1].normal;
        glm::vec3 n2 = activeScene->worldVertices[activeScene->triangles[sampledLight.emitterIndex].v2].normal;

        //  get new ray direction towards selected light source
        glm::vec3 emmisivePoint = Triangle::GetRandomPointOnTriangle(p0, p1, p2, seed);
        glm::vec3 newDir = emmisivePoint - primaryPayload.worldPosition;
        float distance = glm::distance(emmisivePoint, primaryPayload.worldPosition);
        newDir = newDir / distance;
        
        glm::vec3 brdf = MathUtils::CalculateBRDF(
            primaryPayload.worldNormal,
            -primaryRay.direction,
            newDir,
            hitMaterial.albedo,
            hitMaterial.metallic,
            hitMaterial.roughness
        );

        //  rendering equation
        float cosTheta_x = glm::max(glm::dot(newDir, primaryPayload.worldNormal), 0.0f);
        float cosTheta_y = glm::max(glm::dot(-newDir, Triangle::GetTriangleNormal(n0,n1,n2)), 0.0f);
        float triAreaPDF = 1.0f / Triangle::GetTriangleArea(p0,p1,p2);  //  probably could just precompute the triangle's area but that is one more float or two to store per triangle, need to test for memory cost vs performance benefits.
        float totalPDF = sampledLight.pmf * triAreaPDF * (distance * distance);
        
        sampleThroughput *= brdf * cosTheta_x * cosTheta_y / totalPDF;
        
        sampleRay.origin = primaryPayload.worldPosition + primaryPayload.worldNormal * 1e-12f;
        sampleRay.direction = newDir;
        
        samplePayload = TraceRay(sampleRay, activeScene);

        // Miss: hit sky
        if (samplePayload.hitDistance < 0.0f)
        {
            radiance += sampleThroughput * settings.skyColor;
            continue;
        }

        //  check if ray actually hits light source
        if(static_cast<uint32_t>(samplePayload.objectIndex) != sampledLight.emitterIndex)
            continue;   //  if not visible then return no radiance
        
        const Triangle& tri = activeScene->triangles[sampledLight.emitterIndex];
        const Material& material = activeScene->materials[tri.materialIndex];

        // Hit emissive
        glm::vec3 emission = material.GetEmission();
        float emmisiveRadiance = material.GetEmissionRadiance();
        if (emmisiveRadiance > 0.0f)
            radiance += sampleThroughput * emission;
    }

    radiance /= float(sampleCount);
    return glm::vec4(radiance, 1.0f);
}

//  NEXT EVENT ESTIMATION
__host__ __device__ glm::vec4 RendererGPU::PerPixel_NextEventEstimation(
    uint32_t x, uint32_t y,
    uint8_t maxBounces, uint8_t sampleCount,
    uint32_t frameIndex, const RenderingSettings& settings,
    const Scene_GPU* activeScene, const Camera_GPU* activeCamera,
    uint32_t imageWidth)
{
    uint32_t seed = (x + y * imageWidth) * frameIndex;
    glm::vec3 radiance{0.0f};

    // PRIMARY RAY
    Ray ray;
    ray.origin    = activeCamera->position;
    ray.direction = activeCamera->rayDirections[x + y * imageWidth];

    RayHitPayload payload = TraceRay(ray, activeScene);

    //  Skybox
    if (payload.hitDistance < 0.0f)
        return glm::vec4(settings.skyColor, 1.0f);
    
    const Triangle& hitTri = activeScene->triangles[payload.objectIndex];
    const Material& hitMat = activeScene->materials[hitTri.materialIndex];

    // Hit emissive surface
    if (glm::length(hitMat.GetEmission()) > 0.0f)
        return glm::vec4(hitMat.GetEmission(), 1.0f);
    
    // Multi-sample per pixel
    for (int s = 0; s < sampleCount; ++s)
    {
        seed += (s + 1) * 31;   // original seed update

        glm::vec3 pathThroughput{1.0f};
        Ray       pathRay      = ray;
        RayHitPayload hit      = payload;
        
        float pdfBRDF = 1.0f, pdfDirect = 1.0f;
        float weightBRDF = 1.0f, weightDirect = 1.0f;

        // Bounce loop
        for (int bounce = 0; bounce < maxBounces; ++bounce)
        {
            const Triangle& tri = activeScene->triangles[hit.objectIndex];
            const Material& mat = activeScene->materials[tri.materialIndex];
            
            // -------------------------------
            //   DIRECT LIGHT 
            // -------------------------------

            LightTree::ShadingPointQuery sp;
                sp.normal   = hit.worldNormal;
                sp.position = hit.worldPosition;

                LightTree::SampledLight sampled = PickLight_TLAS(activeScene->meshes, activeScene->lightTree_tlas, sp, seed);

                // light triangle data
                const Triangle& lTri = activeScene->triangles[sampled.emitterIndex];
                glm::vec3 p0 = activeScene->worldVertices[lTri.v0].position;
                glm::vec3 p1 = activeScene->worldVertices[lTri.v1].position;
                glm::vec3 p2 = activeScene->worldVertices[lTri.v2].position;
                glm::vec3 n0 = activeScene->worldVertices[lTri.v0].normal;
                glm::vec3 n1 = activeScene->worldVertices[lTri.v1].normal;
                glm::vec3 n2 = activeScene->worldVertices[lTri.v2].normal;

                glm::vec3 lightPoint = Triangle::GetRandomPointOnTriangle(p0,p1,p2,seed);
                glm::vec3 lightDir = lightPoint - hit.worldPosition;
                float dist = glm::length(lightDir);
                lightDir /= dist;

                // Shadow ray
                Ray shadowRay;
                shadowRay.origin = hit.worldPosition + hit.worldNormal * 1e-12f;
                shadowRay.direction = lightDir;

                RayHitPayload shadowPayload = TraceRay(shadowRay, activeScene);

                // Only add if unoccluded and hit correct emitter
                if (shadowPayload.hitDistance > 0.0f &&
                    static_cast<uint32_t>(shadowPayload.objectIndex) == sampled.emitterIndex)
                {
                    glm::vec3 lightNormal = Triangle::GetTriangleNormal(n0,n1,n2);

                    glm::vec3 brdf = MathUtils::CalculateBRDF(
                        hit.worldNormal,
                        -pathRay.direction,
                        lightDir,
                        mat.albedo,
                        mat.metallic,
                        mat.roughness
                    );

                    float cosTheta_x = glm::max(glm::dot(lightDir, hit.worldNormal), 0.0f);
                    float cosTheta_y = glm::max(glm::dot(-lightDir, lightNormal), 1e-12f);

                    // triangle area pdf (area measure)
                    float triArea = Triangle::GetTriangleArea(p0,p1,p2);
                    float triAreaPDF = 1.0f / triArea; // p_A

                    // convert area PDF -> solid-angle PDF:
                    // p_ω = p_A * r^2 / cosTheta_y
                    float lightSolidAnglePDF = triAreaPDF * (dist * dist) / cosTheta_y;

                    // get all the probabilily of directly choosing light source and probabily of choosing that direction according to BRDF
                    pdfDirect = sampled.pmf * lightSolidAnglePDF;
                    pdfBRDF = MathUtils::BRDFHemispherePDF(hit.worldNormal, -pathRay.direction, lightDir, mat.albedo, mat.metallic, mat.roughness);

                    //  Do MIS weighting
                    //  calc balance heuristic
                    weightBRDF = pdfBRDF / glm::max(pdfBRDF + pdfDirect, 1e-12f);
                    weightDirect = 1.0f - weightBRDF;
                    
                    const Material& lightMat = activeScene->materials[lTri.materialIndex];
                    
                    radiance += weightDirect *
                                pathThroughput *
                                brdf *
                                cosTheta_x *
                                lightMat.GetEmission() /
                                pdfDirect;
                }

            //  since not doing GI, dont bother continue
            if(maxBounces == 1)
            {
                radiance /= weightDirect;   //  undo MIS so it is exactly similiar like regular Light Source Sampling
                break;
            }
                
            
            // -------------------------------
            //   INDIRECT BOUNCE
            // -------------------------------
            glm::vec3 nextDir = MathUtils::BRDFSampleHemisphere(
                hit.worldNormal,
                -pathRay.direction,
                mat.albedo,
                mat.metallic,
                mat.roughness,
                seed,
                pdfBRDF 
            );

            pdfBRDF = glm::max(pdfBRDF, 1e-12f);
            
            glm::vec3 brdf = MathUtils::CalculateBRDF(
                hit.worldNormal,
                -pathRay.direction,
                nextDir,
                mat.albedo,
                mat.metallic,
                mat.roughness
            );

            float cosTheta = glm::dot(nextDir, hit.worldNormal);

            // throughput update
            pathThroughput *=  brdf * cosTheta / pdfBRDF;

            pathRay.origin    = hit.worldPosition + hit.worldNormal * 1e-12f;
            pathRay.direction = nextDir;

            // Trace next bounce
            hit = TraceRay(pathRay, activeScene);
            sp.normal   = hit.worldNormal;
            sp.position = hit.worldPosition;
            
            //  if brdf sampling hits light source
            //  Skybox
            if (hit.hitDistance < 0.0f)
            {
                radiance += pathThroughput * settings.skyColor;
                break;
            }

            // Emissive surface
            const Triangle& hitEmissiveTri = activeScene->triangles[hit.objectIndex];
            const Material& hitEmissiveMat = activeScene->materials[hitEmissiveTri.materialIndex];
            glm::vec3 emission = hitEmissiveMat.GetEmission();
            if (hitEmissiveMat.GetEmissionRadiance() > 0.0f)
            {
                glm::vec3 p0 = activeScene->worldVertices[hitEmissiveTri.v0].position;
                glm::vec3 p1 = activeScene->worldVertices[hitEmissiveTri.v1].position;
                glm::vec3 p2 = activeScene->worldVertices[hitEmissiveTri.v2].position;
                glm::vec3 n0 = activeScene->worldVertices[hitEmissiveTri.v0].normal;
                glm::vec3 n1 = activeScene->worldVertices[hitEmissiveTri.v1].normal;
                glm::vec3 n2 = activeScene->worldVertices[hitEmissiveTri.v2].normal;

                glm::vec3 lightPoint = Triangle::GetRandomPointOnTriangle(p0,p1,p2,seed);
                glm::vec3 lightDir = lightPoint - hit.worldPosition;
                float dist = glm::length(lightDir);
                lightDir /= dist;
                glm::vec3 lightNormal = Triangle::GetTriangleNormal(n0,n1,n2);
                float cosTheta_y = glm::max(glm::dot(-lightDir, lightNormal), 1e-12f);
                
                float triArea = Triangle::GetTriangleArea(p0,p1,p2);
                float triAreaPDF = 1.0f / triArea; // p_A

                // convert area PDF -> solid-angle PDF:
                float lightSolidAnglePDF = triAreaPDF * (dist * dist) / cosTheta_y;
                pdfDirect = ComputeDirectEmitterPMF(activeScene->meshes, activeScene->lightTree_tlas, sp, hit.objectIndex);
                pdfDirect *= lightSolidAnglePDF;
                //  Do MIS weighting
                //  calc balance heuristic
                weightBRDF = pdfBRDF / glm::max(pdfBRDF + pdfDirect, 1e-12f);
                radiance += weightBRDF * pathThroughput * emission;
                break;
            }
            
        } // bounce loop
    }     // sample loop

    return glm::vec4(radiance / float(sampleCount), 1.0f);
}

__host__ __device__ RayHitPayload RendererGPU::ClosestHit(
    const Ray& ray, float hitDistance, int objectIndex,
    float u, float v, const Scene_GPU* activeScene)
{
    RayHitPayload payload;
    payload.hitDistance = hitDistance;
    payload.objectIndex = objectIndex;

    const Triangle& tri = activeScene->triangles[objectIndex];
    payload.worldPosition = ray.origin + ray.direction * hitDistance;

    // Interpolate normals
    float w = 1.0f - u - v;
    glm::vec3 n0 = activeScene->worldVertices[tri.v0].normal;
    glm::vec3 n1 = activeScene->worldVertices[tri.v1].normal;
    glm::vec3 n2 = activeScene->worldVertices[tri.v2].normal;

    payload.worldNormal = glm::normalize(n0 * w + n1 * u + n2 * v);

    return payload;
}

__host__ __device__ RayHitPayload RendererGPU::Miss(const Ray& ray)
{
    RayHitPayload payload;
    payload.hitDistance = -1.0f;
    return payload;
}

__global__ void RenderKernel(glm::vec4* accumulationData, uint32_t* renderImageData, uint32_t width, uint32_t height, uint32_t frameIndex, RenderingSettings settings, const Scene_GPU* scene, const Camera_GPU* camera)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    size_t index = x + y * width;

    glm::vec4 pixelColor{0.0f};
    switch(settings.currentSamplingTechnique)
    {
    case BRUTE_FORCE:
        pixelColor = RendererGPU::PerPixel_BruteForce(x, y, static_cast<uint8_t>(settings.lightBounces), static_cast<uint8_t>(settings.sampleCount), frameIndex, settings, scene, camera, width);
        break;
    case UNIFORM_SAMPLING:
        pixelColor = RendererGPU::PerPixel_UniformSampling(x, y, static_cast<uint8_t>(settings.lightBounces), static_cast<uint8_t>(settings.sampleCount), frameIndex, settings, scene, camera, width);
        break;
    case COSINE_WEIGHTED_SAMPLING:
        pixelColor = RendererGPU::PerPixel_CosineWeightedSampling(x, y, static_cast<uint8_t>(settings.lightBounces), static_cast<uint8_t>(settings.sampleCount), frameIndex, settings, scene, camera, width);
        break;
    case GGX_SAMPLING:
        pixelColor = RendererGPU::PerPixel_GGXSampling(x, y, static_cast<uint8_t>(settings.lightBounces), static_cast<uint8_t>(settings.sampleCount), frameIndex, settings, scene, camera, width);
        break;
    case BRDF_SAMPLING:
        pixelColor = RendererGPU::PerPixel_BRDFSampling(x, y, static_cast<uint8_t>(settings.lightBounces), static_cast<uint8_t>(settings.sampleCount), frameIndex, settings, scene, camera, width);
        break;
    case LIGHT_SOURCE_SAMPLING:
        pixelColor = RendererGPU::PerPixel_LightSourceSampling(x, y, static_cast<uint8_t>(settings.lightBounces), static_cast<uint8_t>(settings.sampleCount), frameIndex, settings, scene, camera, width);
        break;
    case NEE:
        pixelColor = RendererGPU::PerPixel_NextEventEstimation(x, y, static_cast<uint8_t>(settings.lightBounces), static_cast<uint8_t>(settings.sampleCount), frameIndex, settings, scene, camera, width);
        break;
    //case RESTIR_DI:
    //case RESTIR_GI:
    default:
        pixelColor = RendererGPU::PerPixel_BruteForce(x, y, static_cast<uint8_t>(settings.lightBounces), static_cast<uint8_t>(settings.sampleCount), frameIndex, settings, scene, camera, width);
    }
    

    
    // Prevent NaNs or Infs from propagating
    if (!glm::all(glm::isfinite(pixelColor)))
        pixelColor = glm::vec4(0.0f);

    // Accumulate pixel color
    accumulationData[index] += pixelColor;

    // Average over frames
    glm::vec4 accumulatedColor = accumulationData[index] / (float)frameIndex;

    // Simple tone mapping for HDR
    accumulatedColor = accumulatedColor / (accumulatedColor + glm::vec4(1,1,1,0));

    // Clamp to valid range
    accumulatedColor = glm::clamp(accumulatedColor, glm::vec4(0.0f), glm::vec4(1.0f));

    // Convert to packed RGBA
    renderImageData[index] = ColorUtils::ConvertToRGBA(accumulatedColor);
}
